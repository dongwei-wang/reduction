#include <stdio.h>
#include "reduction_kernel.cu"

#define UPPER_BOUND 1000
#define BLOCK_SIZE 1024

void runTest();
int* arrayInit(unsigned len);
int computeOnDevice(int* h_data, int array_mem_size);
int computeGold( int* data, const int len);

int main(){
	runTest();
	return 0;
}

void runTest(){
	printf("Please input the number of elements in the array: \n");
	unsigned num_elements=4096;
	/* if( scanf("%d", &num_elements) != 1){ */
	/*     printf("Input Failed\n"); */
	/*     return; */
	/* } */

	int* h_data = arrayInit(num_elements);
	int reference = computeGold(h_data, num_elements);
	int result = computeOnDevice(h_data, num_elements);
	printf( "Test %s !!!\n", (reference == result) ? "PASSED" : "FAILED");
	printf( "Device: %d  Host: %d\n", result, reference);
	free(h_data);
	return;
}

int* arrayInit(unsigned len){
	int* data = (int*) malloc(sizeof(int)*len);
	for( unsigned i=0; i<len; i++ ){
		//data[i] = rand()%UPPER_BOUND;
		data[i] = 1;
	}
	return data;
}

int computeOnDevice(int* h_data, int len){
	int* d_data = NULL;
	unsigned block_cnt = ((len>>1) + BLOCK_SIZE - 1)/BLOCK_SIZE;

	printf("The length is %d\n", len);
	printf("The block  is %d\n", block_cnt );

	dim3 gridDim(block_cnt, 1);
	dim3 blockDim(BLOCK_SIZE, 1);

	hipMalloc((void**)&d_data, len * sizeof(int));
	hipMemcpy(d_data, h_data, len * sizeof(int), hipMemcpyHostToDevice);
	reduction<<<gridDim, blockDim>>>(d_data, len);
	hipMemcpy(h_data, d_data, len * sizeof(int), hipMemcpyDeviceToHost);
	hipFree(d_data);
	return h_data[0];
}

int computeGold( int* data, int len){
	int total_sum = 0;
	for( int i = 0; i < len; ++i){
		total_sum += data[i];
	}
	return total_sum;
}

